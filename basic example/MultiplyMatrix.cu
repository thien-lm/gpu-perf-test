
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    const int N = 4096; // Matrix size (N x N)
    const int blockSize = 16;
    const dim3 gridDims(N / blockSize, N / blockSize);

    // Allocate memory on the host
    float* h_A = new float[N * N];
    float* h_B = new float[N * N];
    float* h_C = new float[N * N];

    // Initialize input matrices
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i);
    }

    // Allocate memory on the device (GPU)
    float* d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    matrixMultiply<<<gridDims, dim3(blockSize, blockSize)>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print a few elements of the result matrix
    for (int i = 0; i < 10; ++i) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
